#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "sudoku.h"

const int block_width = SUB_N;
const int block_height = SUB_N;

void copy(int* host_data, Grid* grid){
    for (int i = 0; i < N * N; i++)
        host_data[i] = (*grid)[i / N][i % N];
}

void test(int* host_data, Grid* grid){
    for (int i = 0; i < N * N; i++){
        printf("%d ", host_data[i]);
        if (i+1 % N == 0) printf("\n");
    }
    printf("====================================\n");
    show_grid(grid);
}

__global__ void cuda_bfs(int* new_boards, int* old_boards, int* empty_spaces, int* empty_space_counts, int* board_index, int total_boards){
    int x_index = blockIdx.x * blockDim.x + threadIdx.x; 
    const int num_tiles = N * N;
    const int offset = N * N * x_index;

    while (x_index < total_boards) {
        // find the next empty spot
        bool found = false;

        for (int i = (x_index * num_tiles); i < (x_index + 1) * num_tiles && !found; i++){
            if (old_boards[i] != UNASSIGNED) continue;
            
            // if (old_board[i] == UNASSIGNED):
            found = true;
            const int real_id = i - offset; 
            const int row = real_id / N;
            const int col = real_id % N;

            // find a suitable value to fill in
            for (int value = 1; value <= N; value++){
                bool can_fill_in = true;

                // fix row, change column
                for (int c = 0; c < N; c++){
                    if (old_boards[row * N + c + offset] == value) { can_fill_in = false; break; }
                }
                if (!can_fill_in) continue;

                // fix column, change row
                for (int r = 0; r < N; r++){
                    if (old_boards[r * N + col + offset] == value) { can_fill_in = false; break; }
                }
                if (!can_fill_in) continue;

                // check box
                // little strange
                for (int r = (row /SUB_N) * SUB_N; r < SUB_N; r++){
                    for (int c = (col / SUB_N) * SUB_N; c < SUB_N; c++){
                        if (old_boards[r * N + c + offset] == value) { can_fill_in = false; break; }
                    } 
                }
                if (!can_fill_in) continue;

                int next_board_index = atomicAdd(board_index, 1);
                int empty_index = 0;
                for (int r = 0; r < N; r++){
                    for (int c = 0; c < N; c++){
                        new_boards[r * N + c + next_board_index * num_tiles] = old_boards[r * N + c + offset];
                        if (old_boards[r * N + c + offset] == UNASSIGNED && (r != row || c != col)){
                            empty_spaces[empty_index + next_board_index * num_tiles] = r * N + c;
                            empty_index++;
                        }
                    }
                }
                empty_space_counts[next_board_index] = empty_index;
                new_boards[row * N + col + next_board_index * num_tiles] = value;
            }
        }
        x_index += blockDim.x * gridDim.x;
    }
}

void hostFE(Grid* grid){
    // allocate init_board and copy grid to it
    int *init_board = (int*) malloc(N * N * sizeof(int));
    copy(init_board, grid);

    // the boards after the next iteration of breadth first search
    int *new_boards;
    // the previous boards, which formthe frontier of the breadth first search
    int *old_boards;
    // stores the location of the empty spaces in the boards
    int *empty_spaces;
    // stores the number of empty spaces in each board
    int *empty_space_count;
    // where to store the next new board generated
    int *board_index;

    // maximum number of boards from breadth first search
    const int num_elements = 1 << 26;    

    // allocate memory
    hipMalloc(&new_boards, num_elements * sizeof(int));
    hipMalloc(&old_boards, num_elements * sizeof(int));
    hipMalloc(&empty_spaces, num_elements * sizeof(int));
    hipMalloc(&empty_space_count, (num_elements / (N * N) + 1) * sizeof(int));
    hipMalloc(&board_index, sizeof(int));
    
    int total_boards = 1;

    // initialize memory
    hipMemset(new_boards, UNASSIGNED, num_elements * sizeof(int));
    hipMemset(old_boards, UNASSIGNED, num_elements * sizeof(int));
    hipMemset(board_index, 0, sizeof(int));

    // copy init_board to old_boards
    hipMemcpy(old_boards, init_board, N * N * sizeof(int), hipMemcpyHostToDevice);

    return;

    const int size = N * N * sizeof(int);
    int* host_data;
    hipHostAlloc(&host_data, size, hipHostMallocMapped);
    copy(host_data, grid);
    //test(host_data, grid);

    size_t pitch;
    int* device_data;
    hipMallocPitch(&device_data, &pitch, N * sizeof(int), N);

    dim3 thread_per_block(block_width, block_height);
    dim3 num_blocks(N / thread_per_block.x, N / thread_per_block.y);

    cuda_solve<<<num_blocks, thread_per_block>>>();

    hipHostFree(host_data);
    hipFree(device_data);
}