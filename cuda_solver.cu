#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "sudoku.h"

#define NUM_TILES N * N
#define n SUB_N

void load(Grid* grid, int *board) {
    for (int i = 0; i < NUM_TILES; i++)
        board[i] = (*grid)[i / N][i % N];
}

void printBoard(int *board) {
    for (int i = 0; i < N; i++) {
        if (i % n == 0) {
            printf("-----------------------\n");
        }

        for (int j = 0; j < N; j++) {
            if (j % n == 0) {
            printf("| ");
            }
            printf("%d ", board[i * N + j]);
        }

        printf("|\n");
    }
    printf("-----------------------\n");
}

/**
 * This kernel has each thread try to solve a different board in the input array using the
 * backtracking algorithm.
 *
 * boards:      This is an array of size numBoards * N * N. Each board is stored contiguously,
 *              and rows are contiguous within the board. So, to access board x, row r, and col c,
 *              use boards[x * N * N + r * N + c]
 *
 * numBoards:   The total number of boards in the boards array.
 *
 * emptySpaces: This is an array of size numBoards * N * N. board is stored contiguously, and stores
 *              the indices of the empty spaces in that board. Note that this N * N pieces may not
 *              be filled.
 *
 * numEmptySpaces:  This is an array of size numBoards. Each value stores the number of empty spaces
 *                  in the corresponding board.
 *
 * finished:    This is a flag that determines if a solution has been found. This is a stopping
 *              condition for the kernel.
 *
 * solved:      This is an output array of size N * N where the solved board is stored.
 */
__global__
void sudokuBacktrack(int *boards,
        const int numBoards,
        int *emptySpaces,
        int *numEmptySpaces,
        int *finished,
        int *solved) {

    int index = blockDim.x * blockIdx.x + threadIdx.x;

    int *currentBoard;
    int *currentEmptySpaces;
    int currentNumEmptySpaces;


    while ((*finished == 0) && (index < numBoards)) {
    
        int emptyIndex = 0;

        currentBoard = boards + index * 81;
        currentEmptySpaces = emptySpaces + index * 81;
        currentNumEmptySpaces = numEmptySpaces[index];

        while ((emptyIndex >= 0) && (emptyIndex < currentNumEmptySpaces)) {

            currentBoard[currentEmptySpaces[emptyIndex]]++;

            if (!validBoard(currentBoard, currentEmptySpaces[emptyIndex])) {

                // if the board is invalid and we tried all numbers here already, backtrack
                // otherwise continue (it will just try the next number in the next iteration)
                if (currentBoard[currentEmptySpaces[emptyIndex]] >= 9) {
                    currentBoard[currentEmptySpaces[emptyIndex]] = 0;
                    emptyIndex--;
                }
            }
            // if valid board, move forward in algorithm
            else {
                emptyIndex++;
            }

        }

        if (emptyIndex == currentNumEmptySpaces) {
            // solved board found
            *finished = 1;

            // copy board to output
            for (int i = 0; i < N * N; i++) {
                solved[i] = currentBoard[i];
            }
        }

        index += gridDim.x * blockDim.x;
    }
}

void cudaSudokuBacktrack(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        int *boards,
        const int numBoards,
        int *emptySpaces,
        int *numEmptySpaces,
        int *finished,
        int *solved) {

    sudokuBacktrack<<<blocks, threadsPerBlock>>>
        (boards, numBoards, emptySpaces, numEmptySpaces, finished, solved);
}

/**
 * This kernel takes a set of old boards and finds all possible next boards by filling in the next
 * empty space.
 *
 * old_boards:      This is an array of size sk. Each N * N section is another board. The rows
 *                  are contiguous within the board. This array stores the previous set of boards.
 *
 * new_boards:      This is an array of size sk. Each N * N section is another board. The rows
 *                  are contiguous within the board. This array stores the next set of boards.
 *
 * total_boards:    Number of old boards.
 *
 * board_index:     Index specifying the index of the next opening in new_boards.
 *
 * empty_spaces:    This is an array of size sk. Each N * N section is another board, storing the
 *                  indices of empty spaces in new_boards.
 *
 * empty_space_count:   This is an array of size sk / N / N + 1 which stores the number of empty
 *                      spaces in the corresponding board.
 */
__global__
void
cudaBFSKernel(int *old_boards,
        int *new_boards,
        int total_boards,
        int *board_index,
        int *empty_spaces,
        int *empty_space_count) {
    
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // board_index must start at zero 

    while (index < total_boards) {
        // find the next empty spot
        int found = 0;

        for (int i = (index * N * N); (i < (index * N * N) + N * N) && (found == 0); i++) {
            // found a open spot
            if (old_boards[i] == 0) {
                found = 1;
                // get the correct row and column shits
                int temp = i - N * N * index;
                int row = temp / N;
                int col = temp % N;
                
                // figure out which numbers work here
                for (int attempt = 1; attempt <= N; attempt++) {
                    int works = 1;
                    // row constraint, test various columns
                    for (int c = 0; c < N; c++) {
                        if (old_boards[row * N + c + N * N * index] == attempt) {
                            works = 0;
                        }
                    }
                    // column contraint, test various rows
                    for (int r = 0; r < N; r++) {
                        if (old_boards[r * N + col + N * N * index] == attempt) {
                            works = 0;
                        }
                    }
                    // box constraint
                    for (int r = n * (row / n); r < n; r++) {
                        for (int c = n * (col / n); c < n; c++) {
                            if (old_boards[r * N + c + N * N * index] == attempt) {
                                works = 0;
                            }
                        }
                    }
                    if (works == 1) {
                        // copy the whole board

                        int next_board_index = atomicAdd(board_index, 1);
                        int empty_index = 0;
                        for (int r = 0; r < 9; r++) {
                            for (int c = 0; c < 9; c++) {
                                new_boards[next_board_index * 81 + r * 9 + c] = old_boards[index * 81 + r * 9 + c];
                                if (old_boards[index * 81 + r * 9 + c] == 0 && (r != row || c != col)) {
                                    empty_spaces[empty_index + 81 * next_board_index] = r * 9 + c;

                                    empty_index++;
                                }
                            }
                        }
                        empty_space_count[next_board_index] = empty_index;
                        new_boards[next_board_index * 81 + row * 9 + col] = attempt;
                    }
                }
            }
        }

        index += blockDim.x * gridDim.x;
    }
}


void callBFSKernel(const unsigned int blocks, 
                        const unsigned int threadsPerBlock,
                        int *old_boards,
                        int *new_boards,
                        int total_boards,
                        int *board_index,
                        int *empty_spaces,
                        int *empty_space_count) {
    cudaBFSKernel<<<blocks, threadsPerBlock>>>
        (old_boards, new_boards, total_boards, board_index, empty_spaces, empty_space_count);
}

void hostFE(Grid* grid){
    const unsigned int threadsPerBlock = N;
    const unsigned int maxBlocks = N; 

    // load the board
    int *board = new int[NUM_TILES];
    load(grid, board);

    // the boards after the next iteration of breadth first search
    int *new_boards;
    // the previous boards, which formthe frontier of the breadth first search
    int *old_boards;
    // stores the location of the empty spaces in the boards
    int *empty_spaces;
    // stores the number of empty spaces in each board
    int *empty_space_count;
    // where to store the next new board generated
    int *board_index;

    // maximum number of boards from breadth first search
    const int sk = 1 << 26;

    // allocate memory on the device
    hipMalloc(&empty_spaces, sk * sizeof(int));
    hipMalloc(&empty_space_count, (sk / NUM_TILES + 1) * sizeof(int));
    hipMalloc(&new_boards, sk * sizeof(int));
    hipMalloc(&old_boards, sk * sizeof(int));
    hipMalloc(&board_index, sizeof(int));

    // same as board index, except we need to set board_index to zero every time and this can stay
    int total_boards = 1;

    // initialize memory
    hipMemset(board_index, 0, sizeof(int));
    hipMemset(new_boards, 0, sk * sizeof(int));
    hipMemset(old_boards, 0, sk * sizeof(int));

    // copy the initial board to the old boards
    hipMemcpy(old_boards, board, N * N * sizeof(int), hipMemcpyHostToDevice);

    // call the kernel to generate boards
    callBFSKernel(maxBlocks, threadsPerBlock, old_boards, new_boards, total_boards, board_index,
        empty_spaces, empty_space_count);

    // number of boards after a call to BFS
    int host_count;
    // number of iterations to run BFS for
    int iterations = 18;

    // loop through BFS iterations to generate more boards deeper in the tree
    for (int i = 0; i < iterations; i++) {
        hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost);

        //printf("total boards after an iteration %d: %d\n", i, host_count);

        hipMemset(board_index, 0, sizeof(int));


        if (i % 2 == 0) {
            callBFSKernel(maxBlocks, threadsPerBlock, new_boards, old_boards, host_count, board_index, empty_spaces, empty_space_count);
        }
        else {
            callBFSKernel(maxBlocks, threadsPerBlock, old_boards, new_boards, host_count, board_index, empty_spaces, empty_space_count);
        }
    }

    hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost);
    //printf("new number of boards retrieved is %d\n", host_count);

    // flag to determine when a solution has been found
    int *dev_finished;
    // output to store solved board in
    int *dev_solved;

    // allocate memory on the device
    hipMalloc(&dev_finished, sizeof(int));
    hipMalloc(&dev_solved, N * N * sizeof(int));

    // initialize memory
    hipMemset(dev_finished, 0, sizeof(int));
    hipMemcpy(dev_solved, board, N * N * sizeof(int), hipMemcpyHostToDevice);

    if (iterations % 2 == 1) {
        // if odd number of iterations run, then send it old boards not new boards;
        new_boards = old_boards;
    }

    cudaSudokuBacktrack(maxBlocks, threadsPerBlock, new_boards, host_count, empty_spaces,
        empty_space_count, dev_finished, dev_solved);


    // copy back the solved board
    int *solved = new int[N * N];

    memset(solved, 0, N * N * sizeof(int));

    hipMemcpy(solved, dev_solved, N * N * sizeof(int), hipMemcpyDeviceToHost);

    printBoard(solved);


    // free memory
    delete[] board;
    delete[] solved;

    hipFree(empty_spaces);
    hipFree(empty_space_count);
    hipFree(new_boards);
    hipFree(old_boards);
    hipFree(board_index);

    hipFree(dev_finished);
    hipFree(dev_solved);
}