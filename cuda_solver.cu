#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "sudoku.h"

// Check whether an insertion is legal
__device__ int cuda_is_safe(const Grid *grid, int row, int col, int num){
    // check is filled or not
    if ((*grid)[row][col] != UNASSIGNED)
        return 0;
    // check row
    // printf("Check Row\n");
    for (int i = 0; i < N; i++)
        if ((*grid)[row][i] == num)
            return 0;
    // check col
    // printf("Check Column\n");
    for (int i = 0; i < N; i++)
        if ((*grid)[i][col] == num)
            return 0;
    // check sub-grid
    // printf("Check Sub-Grid\n");
    int start_row = (row / SUB_N) * SUB_N;
    int start_col = (col / SUB_N) * SUB_N;
    for (int i = 0; i < SUB_N; i++)
        for (int j = 0; j < SUB_N; j++){
            if ((*grid)[start_row + i][start_col + j] == num){
                return 0;
            }
        }
    return 1;
}

// make a single guess: store the guess info (row, col and val wrapped with a cell), current markup and current grid
__device__ inline void cuda_heap_push(Heap *heap, Cell *cell, Markup *markup, Grid *grid){
    heap->cell_arr[heap->count].cell = *cell;
    memcpy(&heap->cell_arr[heap->count].orig_markup, markup, sizeof(Markup));
    memcpy(&heap->cell_arr[heap->count].orig_grid, grid, sizeof(Grid));
    heap->count++;
}

// undo a single guess: pop the top cell and restore originally saved markup and grid
__device__ inline void cuda_heap_pop(Heap *heap, Cell *cell, Markup *markup, Grid *grid){
    heap->count--;
    *cell = heap->cell_arr[heap->count].cell;
    memcpy(markup, &heap->cell_arr[heap->count].orig_markup, sizeof(Markup));
    memcpy(grid, &heap->cell_arr[heap->count].orig_grid, sizeof(Grid));
}

// serialized & non-recursive solver
__global__ void non_recursive_solve(Sudoku *sudoku, int *can_solve){
    Grid *grid = &sudoku->grid;
    Heap *heap = &sudoku->heap;
    Cell top_cell;
    int row = 0, col = 0, val;
    int find_safe_insertion;
    while (row != N)
    {
        find_safe_insertion = 0;
        if ((*grid)[row][col] == UNASSIGNED){
            for (int v = 1; v <= N; v++){
                if (cuda_is_safe(grid, row, col, v)){
                    // find safe insertion
                    Cell cell = {row, col, v};
                    cuda_heap_push(heap, &cell, &sudoku->markup, &sudoku->grid);
                    (*grid)[row][col] = v;
                    find_safe_insertion = 1;
                    break;
                }
            }
            while (!find_safe_insertion){
                if (heap->count == 0){
                    *can_solve = 0;
                    return;
                }
                // fail to find safe insertion, backtrack
                // try other value for the top cell
                cuda_heap_pop(heap, &top_cell, &sudoku->markup, &sudoku->grid);
                row = top_cell.row;
                col = top_cell.col;
                (*grid)[row][col] = UNASSIGNED;
                for (val =  top_cell.val + 1; val <= N; val++)
                {
                    if (cuda_is_safe(grid, row, col, val)){
                        (*grid)[row][col] = val;
                        Cell cell = {row, col, val};
                        cuda_heap_push(heap, &cell, &sudoku->markup, &sudoku->grid);
                        find_safe_insertion = 1;
                        break;
                    }
                }
            }
        }
        row = row + (col == N-1);
        col = (col + 1)%N;
    }
    *can_solve = 1;
}

int hostFE(Sudoku *sudoku){
    int *cuda_can_solve;
    int *can_solve = new int;
    *can_solve = 0;
    hipMalloc(&cuda_can_solve, sizeof(int));
    hipMemset(cuda_can_solve, 0, sizeof(int));

    Sudoku *cuda_sudoku;
    hipMalloc(&cuda_sudoku, sizeof(Sudoku));
    hipMemcpy(cuda_sudoku, sudoku, sizeof(Sudoku), hipMemcpyHostToDevice);

    non_recursive_solve<<<SUB_N, SUB_N>>>(cuda_sudoku, cuda_can_solve);

    hipMemcpy(sudoku, cuda_sudoku, sizeof(Sudoku), hipMemcpyDeviceToHost);
    hipMemcpy(can_solve, cuda_can_solve, sizeof(int), hipMemcpyDeviceToHost);
    return *can_solve; 
}